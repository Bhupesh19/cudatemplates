#include "hip/hip_runtime.h"
/* 
  Cuda Templates.

  Copyright (C) 2008 Institute for Computer Graphics and Vision,
                     Graz University of Technology
  
  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.
  
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>

#include <cudatemplates/copy.hpp>
#include <cudatemplates/devicememorylinear.hpp>
#include <cudatemplates/hostmemoryheap.hpp>
#include <cudatemplates/pack.hpp>


float
frand()
{
  return rand() / (float)RAND_MAX;
}

int
main()
{
  const size_t SIZE = 64;
  const Cuda::Size<2> size(SIZE, SIZE);

  Cuda::HostMemoryHeap<float , 2> h_data1x(size), h_data1y(size), h_data1z(size), h_data1w(size);
  Cuda::HostMemoryHeap<float2, 2> h_data2(size);
  Cuda::HostMemoryHeap<float3, 2> h_data3(size);
  Cuda::HostMemoryHeap<float4, 2> h_data4(size);

  Cuda::DeviceMemoryLinear<float , 2> d_data1x(size), d_data1y(size), d_data1z(size), d_data1w(size);
  Cuda::DeviceMemoryLinear<float2, 2> d_data2(size);
  Cuda::DeviceMemoryLinear<float3, 2> d_data3(size);
  Cuda::DeviceMemoryLinear<float4, 2> d_data4(size);

  for(Cuda::Iterator<2> i = h_data1x.begin(); i != h_data1x.end(); ++i) {
    h_data1x[i] = frand();
    h_data1y[i] = frand();
    h_data1z[i] = frand();
    h_data1w[i] = frand();
  }      

  Cuda::copy(d_data1x, h_data1x);
  Cuda::copy(d_data1y, h_data1y);
  Cuda::copy(d_data1z, h_data1z);
  Cuda::copy(d_data1w, h_data1w);

  Cuda::pack(d_data2, d_data1x, d_data1y);
  Cuda::pack(d_data3, d_data1x, d_data1y, d_data1z);
  Cuda::pack(d_data4, d_data1x, d_data1y, d_data1z, d_data1w);

  Cuda::copy(h_data2, d_data2);
  Cuda::copy(h_data3, d_data3);
  Cuda::copy(h_data4, d_data4);

  for(Cuda::Iterator<2> i = h_data1x.begin(); i != h_data1x.end(); ++i) {
    assert(h_data2[i].x == h_data1x[i]);
    assert(h_data2[i].y == h_data1y[i]);

    assert(h_data3[i].x == h_data1x[i]);
    assert(h_data3[i].y == h_data1y[i]);
    assert(h_data3[i].z == h_data1z[i]);

    assert(h_data4[i].x == h_data1x[i]);
    assert(h_data4[i].y == h_data1y[i]);
    assert(h_data4[i].z == h_data1z[i]);
    assert(h_data4[i].w == h_data1w[i]);
  }      

  return 0;
}
