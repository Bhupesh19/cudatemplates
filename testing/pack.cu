#include "hip/hip_runtime.h"
/* 
  Cuda Templates.

  Copyright (C) 2008 Institute for Computer Graphics and Vision,
                     Graz University of Technology
  
  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.
  
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>

#include <typeinfo>

#include <cudatemplates/copy.hpp>
#include <cudatemplates/devicememorylinear.hpp>
#include <cudatemplates/event.hpp>
#include <cudatemplates/hostmemoryheap.hpp>
#include <cudatemplates/pack.hpp>


const size_t SIZE = 1024;
const int COUNT = 100;


float
frand()
{
  return rand() / (float)RAND_MAX;
}

template <class VectorType, unsigned DataDim>
float
gbps(float ms)
{
  int bytes = sizeof(VectorType) * 2;  // one read plus one write transfer of vector size

  for(int i = DataDim; i--;)
    bytes *= SIZE;

  float gb = bytes / (float)(1 << 30);
  float sec = ms / (1000 * COUNT);
  return gb / sec;
}

template <class ScalarType, unsigned VectorDim, class VectorType, unsigned DataDim>
void
test()
{
  Cuda::Size<DataDim> size;

  for(int i = DataDim; i--;)
    size[i] = SIZE;

  Cuda::HostMemoryHeap<ScalarType, DataDim> h_data_scalar1[VectorDim], h_data_scalar2[VectorDim];
  Cuda::HostMemoryHeap<VectorType, DataDim> h_data_vector(size);
  Cuda::DeviceMemoryLinear<ScalarType, DataDim> d_data_scalar1[VectorDim], d_data_scalar2[VectorDim];
  Cuda::DeviceMemoryLinear<VectorType, DataDim> d_data_vector(size);

  for(int i = VectorDim; i--;) {
    // allocate host memory:
    h_data_scalar1[i].alloc(size);
    h_data_scalar2[i].alloc(size);

    // allocate device memory:
    d_data_scalar1[i].alloc(size);
    d_data_scalar2[i].alloc(size);

    // initialize data:
    for(Cuda::Iterator<DataDim> j = h_data_scalar1[i].begin(); j != h_data_scalar1[i].end(); ++j)
      h_data_scalar1[i][j] = frand();

    // copy data from host to device memory:
    Cuda::copy(d_data_scalar1[i], h_data_scalar1[i]);
  }

  Cuda::Event t0, t1, t2;

  // pack scalars into vector:
  t0.record();
  
  for(int i = COUNT; i--;)
    Cuda::pack(d_data_vector, d_data_scalar1[0], d_data_scalar1[1], d_data_scalar1[2], d_data_scalar1[3]);

  // unpack vector into scalars:
  t1.record();
  
  for(int i = COUNT; i--;)
    Cuda::unpack(d_data_scalar2[0], d_data_scalar2[1], d_data_scalar2[2], d_data_scalar2[3], d_data_vector);

  // report performance:
  t2.record();
  t2.synchronize();
  printf("pack   %dD %s: %f GB / sec\n", VectorDim, typeid(ScalarType).name(), gbps<VectorType, DataDim>(t1 - t0));
  printf("unpack %dD %s: %f GB / sec\n", VectorDim, typeid(ScalarType).name(), gbps<VectorType, DataDim>(t2 - t1));

  // copy data from device to host memory:
  Cuda::copy(h_data_vector, d_data_vector);

  for(int i = VectorDim; i--;)
    Cuda::copy(h_data_scalar2[i], d_data_scalar2[i]);

  // verify packed data:
  for(Cuda::Iterator<DataDim> i = h_data_vector.begin(); i != h_data_vector.end(); ++i) {
    const ScalarType *vec = &h_data_vector[i].x;

    for(int j = VectorDim; j--;) {
      assert(h_data_scalar2[j][i] == h_data_scalar1[j][i]);
      assert(vec[j] == h_data_scalar2[j][i]);
    }
  }
}

int
main()
{
  test<unsigned char, 4, uchar4, 2>();
  test<short, 4, short4, 2>();
  test<int, 4, int4, 2>();
  test<float, 4, float4, 2>();

  return 0;
}
