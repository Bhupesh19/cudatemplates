#include "hip/hip_runtime.h"
/*
  Cuda Templates.

  Copyright (C) 2008 Institute for Computer Graphics and Vision,
                     Graz University of Technology

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#define USE_CUDA30 0


#include <iostream>

#include <GL/gl.h>
#include <GL/glext.h>
#include <GL/glut.h>

#include <cudatemplates/copy_constant.hpp>


typedef struct uchar3 PixelType;

#if USE_CUDA30

#include <cudatemplates/graphics/opengl/buffer.hpp>
typedef Cuda::Graphics::OpenGL::Buffer<PixelType, 2> FramebufferType;

#else

#include <cudatemplates/opengl/bufferobject.hpp>
typedef Cuda::OpenGL::BufferObject<PixelType, 2> FramebufferType;

#endif


using namespace std;


Cuda::Size<2> size0(512, 512);

FramebufferType *bufobj = 0;


void
reshape(int w, int h)
{
  glViewport(0, 0, w, h);
  bufobj->realloc(Cuda::Size<2>(w, h));
}

void
display()
{
  // clear framebuffer:
  Cuda::copy(*bufobj, make_uchar3(255, 255, 128));

  // save OpenGL state and reset transformation:
  glPushAttrib(GL_ENABLE_BIT);
  glDisable(GL_DEPTH_TEST);
  glMatrixMode(GL_PROJECTION);
  glPushMatrix();
  glLoadIdentity();
  glMatrixMode(GL_MODELVIEW);
  glPushMatrix();
  glLoadIdentity();

  // transfer pixels:
  glRasterPos2i(-1, -1);

#if USE_CUDA30
  bufobj->setState(Cuda::Graphics::Resource::STATE_GRAPHICS_BOUND);
#else
  bufobj->disconnect();
  bufobj->bind();
#endif

  glDrawPixels(bufobj->size[0], bufobj->size[1], GL_RGB, GL_UNSIGNED_BYTE, 0);

#if USE_CUDA30
  bufobj->setState(Cuda::Graphics::Resource::STATE_CUDA_MAPPED);
#else
  bufobj->unbind();
  bufobj->connect();
#endif

  // restore OpenGL state:
  glMatrixMode(GL_PROJECTION);
  glPopMatrix();
  glMatrixMode(GL_MODELVIEW);
  glPopMatrix();
  glPopAttrib();

  // postprocess:
  glutSwapBuffers();
  glutPostRedisplay();
}

void
keyboard(unsigned char c, int, int)
{
  if(c == 0x1b)
    exit(0);
}

int
main(int argc, char *argv[])
{
  try {
#if USE_CUDA30
    CUDA_CHECK(cudaGLSetGLDevice(0));
#endif

    // init GLUT:
    glutInit(&argc, argv);
    glutInitWindowSize(size0[0], size0[1]);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
    glutCreateWindow("CUDA render demo");
    glutReshapeFunc(reshape);
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);

    // create OpenGL buffer object:
    bufobj = new FramebufferType(size0, GL_PIXEL_UNPACK_BUFFER, GL_DYNAMIC_COPY);

    // enter main loop:
    glutMainLoop();
  }
  catch(const std::exception &e) {
    cerr << e.what() << endl;
    return 1;
  }

  delete bufobj;
  return 0;
}
